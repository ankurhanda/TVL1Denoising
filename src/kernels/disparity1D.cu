#include "hip/hip_runtime.h"
#ifndef _DISPARITY1D_KERNEL_H_
#define _DISPARITY1D_KERNEL_H_

#include <stdio.h>
#include <cutil_inline.h>

texture<float, 2, hipReadModeElementType> my_tex;


const static hipChannelFormatDesc chandesc_float1 =
hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define Nsize_hf_Width 1


__global__ void disparity_kernel_q(float* dq, float *du,
                                   float *du0,  float sigma_q, float lambda, float *dI1,
                                   unsigned int width, unsigned int height, unsigned int stride)
{

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

//    float xinterp0 = max(0.0f,min(width*1.0,(float)x+ du0[y*stride+x]));
//    float xinterp1 = max(0.0f,min(width*1.0,(float)x+ du0[y*stride+x]+1));

    float xinterp0 = (float)x+ du0[y*stride+x];
    float xinterp1 = (float)x+ du0[y*stride+x]+1;

//    float xinterp1 = max(0.0f,min(width*1.0,(float)x+ du0[y*stride+x]+1));



    float I2_u0      = tex2D(my_tex,xinterp0+0.5,(float)y+0.5);
    float I1_val     = dI1[y*stride+x];
    float grad_I2_u0 = tex2D(my_tex,xinterp1+0.5,(float)y+0.5) - tex2D(my_tex,xinterp0+0.5,(float)y+0.5);

    float u  = du[y*stride+x];
    float u0 = du0[y*stride+x];

//    float data_term  = lambda*(I2_u0 + (u-u0)*grad_I2_u0 - I1_val);
//    float data_term  = lambda*((I2_u0-I1_val)*(I2_u0-I1_val) + (u-u0)*grad_I2_u0*(u-u0)*grad_I2_u0 + 2*(u-u0)*grad_I2_u0*(I2_u0-I1_val));

//    float mu2 = 0, mu1=0, corr = 0;
    int count = 0;
    float gradI2_sqr = 0, I2_u0_minus_I1 =0, I2_u0_minus_I1_times_grad=0;

    for (int i = -Nsize_hf_Width ; i <= Nsize_hf_Width ; i++)
    {
        for (int j = -Nsize_hf_Width ; j <= Nsize_hf_Width ; j++ )
        {

            if ( x+j < width && x-j >= 0 && y+i < height && y-i >= 0 )
            {
                   float xinterp0 = max(0.0f,min(width*1.0,(float)(x+j)+ du0[(y+i)*stride+(x+j)]));
                   float xinterp1 = max(0.0f,min(width*1.0,(float)(x+j)+ du0[(y+i)*stride+(x+j)]+1));

                   float grad_I2_u0 = tex2D(my_tex,xinterp1+0.5,(float)(y+i)+0.5) - tex2D(my_tex,xinterp0+0.5,(float)(y+i)+0.5);


                   gradI2_sqr += grad_I2_u0*grad_I2_u0;
                   I2_u0_minus_I1 += (tex2D(my_tex,xinterp0+0.5,y+i+0.5) - dI1[(y+i)*stride+x+j])*(tex2D(my_tex,xinterp0 + 0.5,y+i+0.5) - dI1[(y+i)*stride+x+j]);
                   I2_u0_minus_I1_times_grad += (tex2D(my_tex,xinterp0+0.5,y+i+0.5) - dI1[(y+i)*stride+(x+j)])*(grad_I2_u0);

//                   mu2 +=   tex2D(my_tex,xinterp0,y+i);
//                   mu1 +=   dI1[(y+i)*stride+(x+j)];
                   count++;
            }

        }

    }

    float data_term  = lambda*( I2_u0_minus_I1 + (u-u0)*(u-u0)*gradI2_sqr + (u-u0)*I2_u0_minus_I1_times_grad) ;

//    mu2 = mu2/(float)count;
//    mu1 = mu1/(float)count;

//    float sum_grad_I2_u0_times_I1 = 0;

//    for (int i = -3 ; i <= 3 ; i++)
//    {
//        for (int j = -3 ; j <= 3 ; j++ )
//        {
//            if ( x + j < width && x-j >=0 && y+i< height && y-i>=0 )
//            {
//                float xinterp0 = max(0.0f,min(width*1.0,(float)(x+j) + du0[(y+i)*stride+x+j]));
//                float xinterp1 = max(0.0f,min(width*1.0,(float)(x+j) + du0[(y+i)*stride+x+j]+1));

//                 corr += (tex2D(my_tex,xinterp0,y+i) - mu2)*(dI1[(y+i)*stride+(x+j)] - mu1);

//                 float grad_I2_u0 = tex2D(my_tex,xinterp1,(float)y) - tex2D(my_tex,xinterp0,(float)y);
//                 sum_grad_I2_u0_times_I1 += grad_I2_u0*(dI1[(y+i)*stride+(x+j)] - mu1);
//            }
//        }
//    }

//    float data_term = 1 - ( corr/count + (u-u0)*sum_grad_I2_u0_times_I1 / count );


    dq[y*stride+x] = dq[y*stride+x] + sigma_q*(data_term);

    // reprojection
    float reprojection_q = max(1.0f,fabs(dq[y*stride+x]));
    dq[y*stride+x] = dq[y*stride+x] / reprojection_q;
}

extern "C" void launch_disparity_kernel_q(float* dq, float *du,
                                          float *du0,  float sigma_q, float lambda,float *dI1,
                                          unsigned int width, unsigned int height, unsigned int stride)
{
    dim3 block(8,8,1);
    dim3 grid(width / block.x, height / block.y, 1);
    disparity_kernel_q<<<grid, block>>>(dq,du,du0,sigma_q,lambda,dI1,width,height,stride);


}

//launch_disparity_kernel_p (px,py,u,width,height,stride,sigma_p);

__global__ void disparity_kernel_p(float* dpx, float *dpy, float *du,
                                   unsigned int width, unsigned int height, unsigned int stride,
                                   float sigma_p)

{
    float u_dx = 0, u_dy = 0;

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ( x + 1 < width )
    {
        u_dx = du[y*stride+(x+1)] - du[y*stride+x];
    }

    if ( y + 1 < height )
    {
        u_dy = du[(y+1)*stride+x] - du[y*stride+x];
    }

    float pxval = dpx[y*stride+x] + sigma_p*(u_dx);
    float pyval = dpy[y*stride+x] + sigma_p*(u_dy);

    // reprojection
    float reprojection_p   = max(1.0f,sqrt(pxval*pxval + pyval*pyval));

    dpx[y*stride+x] = pxval / reprojection_p;
    dpy[y*stride+x] = pyval / reprojection_p;
}

extern "C" void launch_disparity_kernel_p(float* dpx, float *dpy, float *du,
                                          unsigned int width, unsigned int height, unsigned int stride,
                                          float sigma_p)
{
    dim3 block(8,8,1);
    dim3 grid(width / block.x, height / block.y, 1);
    disparity_kernel_p<<<grid, block>>>(dpx, dpy, du, width,height,stride,sigma_p);

}

__global__ void disparity_kernel_u(float* dpx, float* dpy,float *du, float *du0, float *dq, float *dI1,
                                   float sigma_u, float lambda, unsigned int width,
                                   unsigned height, unsigned int stride)
{
    float dxp = 0 , dyp = 0;

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ( x >= 1 && x < width )  dxp = dpx[y*stride+x] - dpx[y*stride+(x-1)];

    if ( y >= 1 && y < height ) dyp = dpy[y*stride+x] - dpy[(y-1)*stride+x];

    float div_p = dxp + dyp;

    float xinterp0 = max(0.0f,min(width,(float)x+ du0[y*stride+x]));
    float xinterp1 = max(0.0f,min(width,(float)x+ du0[y*stride+x]+1));

//    float xinterp0 = (float)x+ du0[y*stride+x];
//    float xinterp1 = (float)x+ du0[y*stride+x]+1;

    float grad_I2_u0 = tex2D(my_tex,xinterp1+0.5,(float)y+0.5) - tex2D(my_tex,xinterp0+0.5,(float)y+0.5);

    float I1_val = dI1[y*stride+x];

//    float diff_term = lambda*dq[y*stride+x]*grad_I2_u0 - div_p;

//    float diff_term = lambda*(2*(du[y*stride+x]-du0[y*stride+x])*(grad_I2_u0)*grad_I2_u0 + 2*grad_I2_u0*(tex2D(my_tex,xinterp0,(float)y) - I1_val)) - div_p;


//    float mu2=0,mu1=0;
    int count = 0;
//    for (int i = -3 ; i <= 3 ; i++)
//    {
//        for (int j = -3 ; j <= 3 ; j++ )
//        {

//            if ( x+j < width && x-j >= 0 && y+i < height && y-i >= 0 )
//            {
//                   float xinterp0 = max(0.0f,min(width*1.0,(float)(x+j)+ du0[(y+i)*stride+(x+j)]));
//                   mu2 += tex2D(my_tex,xinterp0,y+i);
//                   mu1 += dI1[(y+i)*stride+(x+j)];
//                   count++;
//            }

//        }

//    }

//    mu2 = mu2/(float)count;
//    mu1 = mu1/(float)count;

//    float sum_grad_I2_u0_times_I1 = 0;

//    for (int i = -3 ; i <= 3 ; i++)
//    {
//        for (int j = -3 ; j <= 3 ; j++ )
//        {
//            if ( x + j < width && x-j >=0 && y+i< height && y-i>=0 )
//            {
//                float xinterp0 = max(0.0f,min(width*1.0,(float)(x+j)+ du0[(y+i)*stride+x+j]));
//                float xinterp1 = max(0.0f,min(width*1.0,(float)(x+j)+ du0[(y+i)*stride+x+j]+1));
//                float grad_I2_u0 = tex2D(my_tex,xinterp1,(float)y) - tex2D(my_tex,xinterp0,(float)y);

//                sum_grad_I2_u0_times_I1 += grad_I2_u0*(dI1[(y+i)*stride+(x+j)] - mu1);
//            }
//        }
//    }

//    float diff_term = sum_grad_I2_u0_times_I1 / count;


    float gradI2_sqr = 0, I2_u0_minus_I1_times_grad=0;

    for (int i = -Nsize_hf_Width ; i <= Nsize_hf_Width ; i++)
    {
        for (int j = -Nsize_hf_Width ; j <= Nsize_hf_Width ; j++ )
        {

            if ( x+j < width && x-j >= 0 && y+i < height && y-i >= 0 )
            {
                   float xinterp0 = max(0.0f,min(width*1.0,(float)(x+j)+ du0[(y+i)*stride+(x+j)]));
                   float xinterp1 = max(0.0f,min(width*1.0,(float)(x+j)+ du0[(y+i)*stride+(x+j)]+1));

                   float grad_I2_u0 = tex2D(my_tex,xinterp1+0.5,(float)(y+i)+0.5) - tex2D(my_tex,xinterp0+0.5,(float)(y+i)+0.5);

                       gradI2_sqr += grad_I2_u0*grad_I2_u0;
                   I2_u0_minus_I1_times_grad += (tex2D(my_tex,xinterp0+0.5,y+i+0.5) - dI1[(y+i)*stride+(x+j)])*(grad_I2_u0);

//                   mu2 +=   tex2D(my_tex,xinterp0,y+i);
//                   mu1 +=   dI1[(y+i)*stride+(x+j)];
                   count++;
            }

        }

    }

    float diff_term = lambda*(2*(du[y*stride+x]-du0[y*stride+x])*gradI2_sqr+ 2*I2_u0_minus_I1_times_grad) - div_p;


    du[y*stride+x]  = du[y*stride+x] - sigma_u*(diff_term);
}


extern "C" void launch_disparity_kernel_u(float* dpx, float* dpy,float *du, float *du0,float *dq, float *dI1,
                                          float sigma_u, float lambda, unsigned int width,
                                          unsigned int height, unsigned int stride)
{
    dim3 block(8,8,1);
    dim3 grid(width / block.x, height / block.y, 1);
    disparity_kernel_u<<<grid,block>>>(dpx,dpy,du,du0,dq,dI1,sigma_u,lambda,width,height,stride);

}

extern "C" void launch_disparity_kernel(float* d_I2, /*float* dq, float *u,
                                        float* u0,   float* px, float *py,*/
                                        unsigned int width, unsigned int height, unsigned int imgStride
                                        /*, float sigma_p, float sigma_q, float lambda*/)
{
    dim3 block(8,8,1);
    dim3 grid(width / block.x, height / block.y, 1);

    hipBindTexture2D(0,my_tex,d_I2,chandesc_float1,width, height,imgStride*sizeof(float));
    my_tex.addressMode[0] = hipAddressModeClamp;
    my_tex.addressMode[1] = hipAddressModeClamp;
    my_tex.filterMode = hipFilterModeLinear;
    my_tex.normalized = false;    // access with normalized texture coordinates


//    kernel_disparity_estimation<<< grid, block>>>(u, u0, p, q, I1,sigma_q, sigma_p, sigma_u, lambda);
}

__global__ void disparity_kernel_copy_u0_to_u(float* du, float *du0,
                                   unsigned int width, unsigned int height, unsigned int stride)
{

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    du0[y*stride+x] = du[y*stride+x];
}

extern "C" void  launch_disparity_kernel_copy_u0_to_u(float *du,float *du0, unsigned int width, unsigned int height, unsigned int stride)
{
    dim3 block(8,8,1);
    dim3 grid(width / block.x, height / block.y, 1);
    disparity_kernel_copy_u0_to_u<<<grid,block>>>(du, du0, width,height,stride);
}

__global__ void disparity_kernel_I2warped(float *dI2warped,float *du, unsigned int width, unsigned int height, unsigned int stride)
{

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float xinterp0 = max(0,min(width*1.0,(float)x+ du[y*stride+x]));
//    float xinterp1 = max(0,min(width*1.0,(float)x+ du0[y*stride+x]+1));


    dI2warped[y*stride+x] = tex2D(my_tex,xinterp0+0.5,y+0.5);
}

extern "C" void launch_disparity_kernel_I2warped(float *dI2warped,float *du, unsigned int width, unsigned int height, unsigned int stride)
{
    dim3 block(8,8,1);
    dim3 grid(width / block.x, height / block.y, 1);
    disparity_kernel_I2warped<<<grid,block>>>(dI2warped, du, width,height,stride);
}




#endif // #ifndef _DISPARITY1D_KERNEL_H_
