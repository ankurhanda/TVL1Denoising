#include "hip/hip_runtime.h"
#define HAVE_TOON
#undef isfinite
#undef isnan

#include <math.h>
#include <TooN/TooN.h>
#include <TooN/Cholesky.h>
#include <TooN/LU.h>
#include <boost/math/common_factor.hpp>

#include <stdio.h>
#include <cutil_inline.h>
#include "cumath.h"
#include "primal_dual_update.h"
#include <iostream>
#include <thrust/sort.h>
#include <thrust/pair.h>


texture<float, 2, hipReadModeElementType> TexImgCur;

const static hipChannelFormatDesc chandesc_float1 =
hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

texture<float, 3, hipReadModeElementType> TexImgStack;



__global__ void kernel_doOneIterationUpdatePrimal ( float* d_u,
                                                   const float* d_u0,
                                                   const unsigned int stride,
                                                   const unsigned int width,
                                                   const unsigned int height,
                                                   const float* d_data_term,
                                                   const float* d_gradient_term,
                                                   const float* d_px,
                                                   const float* d_py,
                                                   const float* d_q,
                                                   const float lambda,
                                                   const float sigma_u,
                                                   const float sigma_q,
                                                   const float sigma_p,
                                                   const int _nimages)
{

    /// Update Equations should be
    /// u = u - tau*( lambda*q*grad - divp )

    float dxp = 0 , dyp = 0;

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ( x >= 1 && x < width  )  dxp = d_px[y*stride+x] - d_px[y*stride+(x-1)];

    if ( y >= 1 && y < height )  dyp = d_py[y*stride+x] - d_py[(y-1)*stride+x];

    float div_p = dxp + dyp;

//    float u_update = d_u[y*stride+x] + sigma_u*div_p - sigma_u*lambda*d_q[y*stride+x]*d_gradient_term[y*stride+x];

//    d_u[y*stride+x] = u_update;

//    float grad_sqr = d_gradient_term[y*stride+x]*d_gradient_term[y*stride+x];

//    float u_ = (d_u[y*stride+x] + sigma_u*(div_p));

//    float u0 = d_u0[y*stride+x];

//    float rho = d_data_term[y*stride+x] + (u_-u0)*d_gradient_term[y*stride+x];

//    if ( rho < -sigma_u*lambda*grad_sqr)

//        d_u[y*stride+x] =  u_ + sigma_u*lambda*d_gradient_term[y*stride+x];

//    else if( rho > sigma_u*lambda*grad_sqr)

//        d_u[y*stride+x] =  u_ - sigma_u*lambda*d_gradient_term[y*stride+x];

//    else if ( fabs(rho) <= sigma_u*lambda*grad_sqr)
//        d_u[y*stride+x] =  u_ - rho/(d_gradient_term[y*stride+x]+10E-6);



    /// Have a confusion of _nimages

//    float u0  = d_u0[y*stride+x];
//    float u_  = d_u[y*stride+x] + sigma_u*(div_p);
//    float ai, bi, ti;

//    float *ti_vals = new float [_nimages-1];
//    unsigned int *ti_indices = new unsigned int [_nimages-1];
//    float sum_all_grads = 0;

//    for(int i = 0 ; i < _nimages -1 ; i++)
//    {

//        bi = d_data_term[y*stride+x+i*slice_stride] - u0*d_gradient_term[y*stride+x+i*slice_stride];
//        ai = d_gradient_term[y*stride+x+i*slice_stride];
//        ti = -bi/(ai+1E-6);

//        ti_vals[i]=ti;
//        ti_indices[i]=i;

//        sum_all_grads += ai;
//    }

    /// Sort them.
//    thrust::stable_sort_by_key(ti_vals,ti_vals+_nimages-1,ti_indices);



    /// Find if this lies in between any of these consecutive ti s.
//    float sum_grads_less_k = 0;
//    float sum_grads_great_k = sum_all_grads;
//    int index = 0;
//    bool found_min = false;
//    float rho=0;

//    for(int i = 1 ; i <_nimages - 1 ; i++)
//    {
//        index = ti_indices[i];

//        rho = d_data_term[y*stride+x+index*slice_stride] + (u_-u0)*d_gradient_term[y*stride+x+index*slice_stride];

//        sum_grads_less_k  += d_gradient_term[y*stride+x+index*slice_stride];
//        sum_grads_great_k -= sum_grads_less_k;

//        if (  rho < sigma_u*lambda*(sum_grads_less_k- sum_grads_great_k) && rho > d_gradient_term[y*stride+x+index*slice_stride]*(ti_vals[i-1] - ti_vals[i]) )
//        {
//            d_u[y*stride] = u_ + sigma_u*lambda*(sum_grads_less_k - sum_grads_great_k);
//            found_min = true;
//            return;
//        }
//    }

//    if ( !found_min )
//    {
//        /// Bound check at 0
//        index = ti_indices[0];
//        rho = d_data_term[y*stride+index*slice_stride] + (u_-u0)*d_gradient_term[y*stride+x+index*slice_stride];
//        if ( rho < -sigma_u*lambda*sum_all_grads)
//        {
//            d_u[y*stride+x]  = u_ +  sigma_u*lambda*sum_all_grads;
//            return;
//        }

//        /// Bound check at last
//        index = ti_indices[_nimages-2];
//        rho = d_data_term[y*stride+index*slice_stride] + (u_-u0)*d_gradient_term[y*stride+x+index*slice_stride];
//        if ( rho > sigma_u*lambda*sum_all_grads)
//        {
//            d_u[y*stride+x]  = u_ -  sigma_u*lambda*sum_all_grads;
//            return;
//        }


//        /// Check for minima among the ti points
//        float cur_min_cost = 1E20;
//        int min_di_index = 0;

//        for(int i = 0 ; i < _nimages-1 ; i++)
//        {
//            float di = ti_vals[i];
//            float sum_rhos_at_di = 0;

//            for(int j = 0 ; j < _nimages -1 ; j++)
//            {
//                bi = d_data_term[y*stride+x+j*slice_stride] - u0*d_gradient_term[y*stride+x+j*slice_stride];
//                ai = d_gradient_term[y*stride+x+j*slice_stride];

//                sum_rhos_at_di += fabs(ai*di-bi);
//            }

//           float min_cost = -div_p * di + lambda * sum_rhos_at_di;

//           if ( min_cost < cur_min_cost)
//           {
//               cur_min_cost = min_cost;
//               min_di_index = i;
//           }

//        }

//        float rho_u = d_data_term[y*stride+x+min_di_index*slice_stride] + (u_- u0)*d_gradient_term[y*stride+x+min_di_index*slice_stride];
//        float gradient_at_u = d_gradient_term[y*stride+x+min_di_index*slice_stride];

//        d_u[y*stride+x] = u_ - rho_u/(gradient_at_u+1E-6);

//        return;

//    }

//        d_u[y*stride+x] = fmaxf(1E-6,fminf(1.0f,d_u[y*stride+x]));
//        float diff_term = d_q[y*stride+x]*d_gradient_term[y*stride+x] - div_p;




}

void  doOneIterationUpdatePrimal ( float* d_u,
                                  const float* d_u0,
                                 const unsigned int stride,
                                 const unsigned int width,
                                 const unsigned int height,
                                 const float* d_data_term,
                                 const float* d_gradient_term,
                                 const float* d_px,
                                 const float* d_py,
                                 const float* d_q,
                                 const float lambda,
                                 const float sigma_u,
                                 const float sigma_q,
                                 const float sigma_p,
                                 const int _nimages)
{

    dim3 block(boost::math::gcd<unsigned>(width,32), boost::math::gcd<unsigned>(height,32), 1);
    dim3 grid( width / block.x, height / block.y);

    kernel_doOneIterationUpdatePrimal<<<grid,block>>>( d_u,
                                                       d_u0,
                                                       stride,
                                                       width,
                                                       height,
                                                       d_data_term,
                                                       d_gradient_term,
                                                       d_px,
                                                       d_py,
                                                       d_q,
                                                       lambda,
                                                       sigma_u,
                                                       sigma_q,
                                                       sigma_p,
                                                      _nimages);


}




__global__ void kernel_doOneIterationUpdateDualData( float* d_q,
                                             const unsigned int stride,
                                             const unsigned int width,
                                             const unsigned int height,
                                             const float* d_data_term,
                                             const float* d_gradient_term,
                                             float* d_u,
                                             float* d_u0,
                                             const float lambda,
                                             const float sigma_u,
                                             const float sigma_q,
                                             const float sigma_p)
{

    /// Update Equations should be
    /// q = q - sigma_q*( lambda*data_term )
    /// q = q / max(1.0f,fabs(q))

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

//    float epsilon=0.001;
    float u = d_u[y*stride+x];
    float u0 = d_u0[y*stride+x];

    float q_update = d_q[y*stride+x] + sigma_q*lambda*(d_data_term[y*stride+x]+ (u - u0)*d_gradient_term[y*stride+x]);

//    q_update = q_update/(1+epsilon*sigma_q);

    float reprojection_q = max(1.0f,fabs(q_update));

    d_q[y*stride+x] = q_update/reprojection_q;



}



void doOneIterationUpdateDualData( float* d_q,
                                  const unsigned int stride,
                                  const unsigned int width,
                                  const unsigned int height,
                                  const float* d_data_term,
                                  const float* d_gradient_term,
                                  float* d_u,
                                  float* d_u0,
                                  const float lambda,
                                  const float sigma_u,
                                  const float sigma_q,
                                  const float sigma_p)
{

    dim3 block(boost::math::gcd<unsigned>(width,32), boost::math::gcd<unsigned>(height,32), 1);
    dim3 grid( width / block.x, height / block.y);

    kernel_doOneIterationUpdateDualData<<<grid,block>>>(d_q,
                                                       stride,
                                                       width,
                                                       height,
                                                       d_data_term,
                                                       d_gradient_term,
                                                       d_u,
                                                       d_u0,
                                                       lambda,
                                                       sigma_u,
                                                       sigma_q,
                                                       sigma_p);

}







__global__ void kernel_doOneIterationUpdateDualReg (float* d_px,
                                                    float* d_py,
                                                    float* d_u,
                                                    const unsigned int stride,
                                                    const unsigned int width,
                                                    const unsigned int height,
                                                    const float lambda,
                                                    const float sigma_u,
                                                    const float sigma_q,
                                                    const float sigma_p)
{


    /// Update Equations should be
    /// p = p - sigma_p*( grad_d )
    /// p = p / max(1.0f,length(p))

    float u_dx = 0, u_dy = 0;

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ( x + 1 < width )
    {
        u_dx = d_u[y*stride+(x+1)] - d_u[y*stride+x];
    }

    if ( y + 1 < height )
    {
        u_dy = d_u[(y+1)*stride+x] - d_u[y*stride+x];
    }

    float pxval = d_px[y*stride+x] + sigma_p*(u_dx);
    float pyval = d_py[y*stride+x] + sigma_p*(u_dy);

    // reprojection
    float reprojection_p   = fmaxf(1.0f, length( make_float2(pxval,pyval) ) );

    d_px[y*stride+x] = pxval/reprojection_p;
    d_py[y*stride+x] = pyval/reprojection_p;


}



void doOneIterationUpdateDualReg (float* d_px,
                                  float* d_py,
                                  float* d_u,
                                  const unsigned int stride,
                                  const unsigned int width,
                                  const unsigned int height,
                                  const float lambda,
                                  const float sigma_u,
                                  const float sigma_q,
                                  const float sigma_p)
{

    dim3 block(boost::math::gcd<unsigned>(width,32), boost::math::gcd<unsigned>(height,32), 1);
    dim3 grid( width / block.x, height / block.y);

    kernel_doOneIterationUpdateDualReg<<<grid,block>>>(d_px,
                                                       d_py,
                                                       d_u,
                                                       stride,
                                                       width,
                                                       height,
                                                       lambda,
                                                       sigma_u,
                                                       sigma_q,
                                                       sigma_p);

}



__global__ void kernel_computeImageGradient_wrt_depth(const float2 fl,
                                               const float2 pp,
                                               float* d_u,
                                               float* d_u0,
                                               float* d_data_term,
                                               float* d_gradient_term,
                                               cumat<3,3>R,
                                               cumat<3,1>t,
                                               const unsigned int stride,
                                               float* d_ref_img,
                                               const unsigned int width,
                                               const unsigned int height,
                                               bool disparity,
                                               float dmin,
                                               float dmax,
                                               const unsigned int which_image,
                                               const unsigned int slice_stride)
{

    if ( disparity)

    {
        unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;


        float xinterp0 = (float)x+ d_u0[y*stride+x];
        float xinterp1 = (float)x+ d_u0[y*stride+x]+1;


        float I2_u0      = tex2D(TexImgCur,xinterp0+0.5,(float)y+0.5);
        float I1_val     = d_ref_img[y*stride+x];
        float grad_I2_u0 = tex2D(TexImgCur,xinterp1+0.5,(float)y+0.5) - tex2D(TexImgCur,xinterp0+0.5,(float)y+0.5);

//        float u0 = d_u0[y*stride+x];
//        float u  = d_u[y*stride+x];

        d_gradient_term[y*stride+x] = grad_I2_u0 +10E-6;

        float data_term_value  = (I2_u0 /*+ (u-u0)*grad_I2_u0 */- I1_val);

        d_data_term[y*stride+x] = data_term_value;
    }

    else
    {

        unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

        /// Checked!
        float2 invfl = 1.0f/fl;

        /// Checked!
        float3 uvnorm =  make_float3( (x-pp.x)*invfl.x, (y-pp.y)*invfl.y,1);//*(dmax-dmin);
        cumat<3,1> uvnormMat = {uvnorm.x, uvnorm.y, uvnorm.z};


        float zLinearised = d_u0[y*stride+x];
        zLinearised = fmaxf(1E-6,fminf(1.0f,zLinearised));

        cumat<3,1> p3d_r       = {uvnorm.x*zLinearised, uvnorm.y*zLinearised, zLinearised};

        /// Are we really sure of this?
        cumat<3,1> p3d_dest  =  R*p3d_r + t;

        p3d_dest(2,0) = fmax(0.0f,fmin(1.0f,p3d_dest(2,0)));

        float dIdz;
        float Id_minus_Ir;

        float3 p3d_dest_vec = {p3d_dest(0,0), p3d_dest(1,0), p3d_dest(2,0)};

        float2 p2D_live = {p3d_dest(0,0)/p3d_dest(2,0) , p3d_dest(1,0)/p3d_dest(2,0)};

        p2D_live.x= fmaxf(0,fminf(width, p2D_live.x*fl.x + pp.x));
        p2D_live.y= fmaxf(0,fminf(height,p2D_live.y*fl.y + pp.y));



        float Ir =  d_ref_img[y*stride+x];


//        float Id =  tex2D(TexImgCur,  p2D_live.x+0.5f,p2D_live.y+0.5f);
//        float Idx = tex2D(TexImgCur,  p2D_live.x+0.5f+1.0f,p2D_live.y+0.5f);

        float Id =  tex3D(TexImgStack, p2D_live.x+0.5f,     p2D_live.y+0.5f,which_image);
        float Idx = tex3D(TexImgStack, p2D_live.x+0.5f+1.0f,p2D_live.y+0.5f,which_image);


//        if ( p2D_live.x+0.5+1 > (float) width)
//            Idx = Id;

//        float Idy = tex2D(TexImgCur,  p2D_live.x+0.5f,p2D_live.y+0.5f+1.0f);
        float Idy = tex3D(TexImgStack,  p2D_live.x+0.5f,p2D_live.y+0.5f+1.0f, which_image);

//        if ( p2D_live.y+0.5+1 > (float) height)
//            Idy = Id;


        float2 dIdx = make_float2(Idx-Id, Idy-Id);

        p3d_dest_vec.z = p3d_dest_vec.z + 10E-6;

//        float3 dpi_u = make_float3(1/p3d_dest_vec.z, 0,-(p3d_dest_vec.x)/(p3d_dest_vec.z*p3d_dest_vec.z));
//        float3 dpi_v = make_float3(0, 1/p3d_dest_vec.z,-(p3d_dest_vec.y)/(p3d_dest_vec.z*p3d_dest_vec.z));

        float3 dpi_u = make_float3(fl.x/p3d_dest_vec.z, 0,-(fl.x*p3d_dest_vec.x)/(p3d_dest_vec.z*p3d_dest_vec.z));
        float3 dpi_v = make_float3(0, fl.y/p3d_dest_vec.z,-(fl.y*p3d_dest_vec.y)/(p3d_dest_vec.z*p3d_dest_vec.z));

        cumat<3,1> dXdz = R*uvnormMat; ///(-zLinearised*zLinearised);

        float3 dXdz_vec = {dXdz(0,0),dXdz(1,0),dXdz(2,0)};

        dIdz =  dot(dIdx, make_float2( dot(dXdz_vec,dpi_u),  dot(dXdz_vec,dpi_v) ) );

        Id_minus_Ir = Id-Ir;

        d_data_term[y*stride+x + which_image*slice_stride] = Id_minus_Ir ;//+ (u-u0)*dIdz;
        d_gradient_term[y*stride+x + which_image*slice_stride] = dIdz;

    }

}


void doComputeImageGradient_wrt_depth(const float2 fl,
                                    const float2 pp,
                                    float* d_u,
                                    float* d_u0,
                                    float* d_data_term,
                                    float* d_gradient_term,
                                    TooN::Matrix<3,3>R_lr_,
                                    TooN::Matrix<3,1>t_lr_,
                                    const unsigned int stride,
                                    float* d_ref_img,
                                    const unsigned int width,
                                    const unsigned int height,
                                    bool disparity,
                                    float dmin,
                                    float dmax,
                                    const unsigned int which_image,
                                    const unsigned int slice_stride)
{

    dim3 block(boost::math::gcd<unsigned>(width,32), boost::math::gcd<unsigned>(height,32), 1);
    dim3 grid( width / block.x, height / block.y);

    cumat<3,3> R = cumat_from<3,3,float>(R_lr_);
    cumat<3,1> t = cumat_from<3,1,float>(t_lr_);

    kernel_computeImageGradient_wrt_depth<<<grid,block>>>(fl,
                                          pp,
                                          d_u,
                                          d_u0,
                                          d_data_term,
                                          d_gradient_term,
                                          R,
                                          t,
                                          stride,
                                          d_ref_img,
                                          width,
                                          height,
                                          disparity,
                                          dmin,
                                          dmax,
                                          which_image,
                                          slice_stride);
}








__global__ void kernel_doImageWarping( const float2 fl,
                                       const float2 pp,
                                       const cumat<3,3> R,
                                       const cumat<3,1> t,
                                       float* d_cur2ref_warped,
                                       float* d_u,
                                       const unsigned int stride,
                                       const unsigned int width,
                                       const unsigned int height,
                                       bool disparity)
{


    if ( disparity )
    {
        unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

        float xinterp0 = (float)x+ d_u[y*stride+x];


        d_cur2ref_warped[y*stride+x] = tex2D(TexImgCur,xinterp0+0.5,y);
    }

    else
    {
        unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

        float2 invfl = 1.0f/fl;
        float3 uvnorm =  make_float3( (x-pp.x)*invfl.x, (y-pp.y)*invfl.y,1);

        float zLinearised = d_u[y*stride+x];

        cumat<3,1> p3d      = {uvnorm.x*zLinearised, uvnorm.y*zLinearised, uvnorm.z*zLinearised};
        cumat<3,1> p3d_dest = R*p3d + t;
        float2 p2D_live     = {p3d_dest(0,0)/p3d_dest(2,0) , p3d_dest(1,0)/p3d_dest(2,0)};

        p2D_live.x = p2D_live.x*fl.x + pp.x;
        p2D_live.y = p2D_live.y*fl.y + pp.y;

        d_cur2ref_warped[y*stride+x] = tex2D(TexImgCur,p2D_live.x,p2D_live.y);
    }
}



void doImageWarping(const float2 fl,
                    const float2 pp,
                    TooN::Matrix<3,3> R_lr_,
                    TooN::Matrix<3,1> t_lr_,
                    float *d_cur2ref_warped,
                    float *d_u,
                    const unsigned int stride,
                    const unsigned int width,
                    const unsigned int height,
                    bool disparity)
{

    dim3 block(boost::math::gcd<unsigned>(width,32), boost::math::gcd<unsigned>(height,32), 1);
    dim3 grid( width / block.x, height / block.y);

    cumat<3,3> R = cumat_from<3,3,float>(R_lr_);
    cumat<3,1> t = cumat_from<3,1,float>(t_lr_);

    kernel_doImageWarping<<<grid,block>>>(fl,
                                          pp,
                                          R,
                                          t,
                                          d_cur2ref_warped,
                                          d_u,
                                          stride,
                                          width,
                                          height,
                                          disparity);

}





void BindDepthTexture(float* cur_img,
                      unsigned int width,
                      unsigned int height,
                      unsigned int imgStride)

{
    hipBindTexture2D(0,TexImgCur,cur_img,chandesc_float1,width,height,imgStride*sizeof(float));

    TexImgCur.addressMode[0] = hipAddressModeClamp;
    TexImgCur.addressMode[1] = hipAddressModeClamp;
    TexImgCur.filterMode = hipFilterModeLinear;
    TexImgCur.normalized = false;    // access with normalized texture coordinates
}


void BindDataImageStack ( const hipArray *d_volumeArray,
                          const unsigned int width,
                          const unsigned int height,
                          const unsigned int depth,
                          hipChannelFormatDesc channelDesc)
{
    /// Bind array to 3D texture
    cutilSafeCall(hipBindTextureToArray(TexImgStack, d_volumeArray, channelDesc));

    /// Set Texture Parameters
    TexImgStack.normalized = false;                      // Access with normalized texture coordinates
    TexImgStack.filterMode = hipFilterModeLinear;       // Linear interpolation
    TexImgStack.addressMode[0] = hipAddressModeClamp;   // Clamp texture coordinates
    TexImgStack.addressMode[1] = hipAddressModeClamp;
    TexImgStack.addressMode[2] = hipAddressModeClamp;

}




