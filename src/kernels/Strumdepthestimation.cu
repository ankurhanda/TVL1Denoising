#include "hip/hip_runtime.h"


#define HAVE_TOON
#undef isfinite
#undef isnan

#include <math.h>
#include <TooN/TooN.h>
#include <TooN/Cholesky.h>
#include <TooN/LU.h>
#include <boost/math/common_factor.hpp>

#include <stdio.h>
#include <cutil_inline.h>
#include "cumath.h"
#include "Strumdepthestimation.h"
#include <iostream>


#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

texture<float, 2, hipReadModeElementType> TexImg2;

const static hipChannelFormatDesc chandesc_float1 =
hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);


__global__ void depth_estimation_kernel_q (float* dq, float *du, float *du0,  float sigma_q, float lambda, float *dI1,
                                           unsigned int width, unsigned int height, unsigned int stride,
                                           float *data_term)
{

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;


    float qVal = dq[y*stride+x] + sigma_q*lambda*data_term[y*stride+x];

    float reprojection_q = max(1.0f,fabs(qVal));

    dq[y*stride+x] = qVal/reprojection_q;


}

void launch_depth_estimation_kernel_q (float* dq, float *du, float *du0,  float sigma_q, float lambda, float *dI1,
                                       unsigned int width, unsigned int height, unsigned int stride, float *data_term)
{
    dim3 block(8,8,1);
    dim3 grid(width / block.x, height / block.y, 1);

    depth_estimation_kernel_q <<<grid,block>>> (dq,du,du0,sigma_q,
                                                lambda,dI1,width,
                                                height,stride,
                                                data_term);
}



__global__ void depth_estimation_kernel_u(float* dpx, float* dpy,float *du, float *du0, float *dq, float *dI1,
                                   float sigma_u, float lambda, unsigned int width,
                                   unsigned height, unsigned int stride, /*const cumat3x3 KRKinv, cumat<3,1> cudaKt,*/
                                         float dmin, float dmax, float *grad_wrt_d_at_d0)
{
    float dxp = 0 , dyp = 0;

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ( x >= 1 && x < width )  dxp = dpx[y*stride+x] - dpx[y*stride+(x-1)];

    if ( y >= 1 && y < height ) dyp = dpy[y*stride+x] - dpy[(y-1)*stride+x];

    float div_p = dxp + dyp;

    float diff_term = lambda*dq[y*stride+x]*grad_wrt_d_at_d0[y*stride+x] - div_p;

    du[y*stride+x] = du[y*stride+x] - sigma_u*(diff_term);
//    du[y*stride+x] = min(1.0f,max(0.0f,du[y*stride+x]));

}


void launch_depth_estimation_kernel_u(float* dpx, float* dpy,float *du, float *du0,float *dq, float *dI1,
                               float sigma_u, float lambda, unsigned int width,
                               unsigned int height, unsigned int stride, /* const TooN::Matrix<3,3>& K,
                               const TooN::Matrix<3,3>& Kinv, const TooN::Matrix<3,3>& R_lr, const TooN::Matrix<3,1>& Kt,*/
                                      float dmin, float dmax, float *grad_wrt_d_at_d0)
{
    dim3 block(8,8,1);
    dim3 grid(width / block.x, height / block.y, 1);

    depth_estimation_kernel_u<<<grid,block>>>(dpx,dpy,du,du0,dq,dI1,sigma_u,lambda,width,height,stride,
                                              dmin, dmax,
                                              grad_wrt_d_at_d0);

}


__global__ void depth_estimation_kernel_p(float* dpx, float *dpy, float *du,
                                   unsigned int width, unsigned int height, unsigned int stride,
                                   float sigma_p, float dmin, float dmax)

{
    float u_dx = 0, u_dy = 0;

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ( x + 1 < width )
    {
        u_dx = du[y*stride+(x+1)] - du[y*stride+x];
    }

    if ( y + 1 < height )
    {
        u_dy = du[(y+1)*stride+x] - du[y*stride+x];
    }

    float pxval = dpx[y*stride+x] + sigma_p*(u_dx);
    float pyval = dpy[y*stride+x] + sigma_p*(u_dy);

    // reprojection
    float reprojection_p   = max(1.0f,sqrt(pxval*pxval + pyval*pyval));

    dpx[y*stride+x] = pxval / reprojection_p;
    dpy[y*stride+x] = pyval / reprojection_p;
}


void launch_depth_estimation_kernel_p(float* dpx, float *dpy, float *du,
                                          unsigned int width, unsigned int height, unsigned int stride,
                                          float sigma_p, float dmin, float dmax)
{
    dim3 block(8,8,1);
    dim3 grid(width / block.x, height / block.y, 1);
    depth_estimation_kernel_p<<<grid, block>>>(dpx, dpy, du, width,height,stride,sigma_p, dmin, dmax);

}



__global__ void depth_estimation_I2warped(float* dI2warped,float* du,unsigned int width,
                                      unsigned int height,unsigned int stride,const cumat3x3 KRKinv, cumat<3,1> cudaKt,
                                          float dmin,
                                          float dmax)
{

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float uval = du[y*stride+x];//dmin + du[y*stride+x]*(dmax-dmin);

    const cumat<3,1> p0 = {x*uval, y*uval,uval};
    const cumat<3,1> p2 = KRKinv*p0 + cudaKt;
    const float2 pIn2   = {p2(0,0)/p2(2,0), p2(1,0)/p2(2,0)};

    dI2warped[y*stride+x] = tex2D(TexImg2,pIn2.x+0.5,pIn2.y+0.5);

}


void launch_depth_estimation_kernel_I2warped(float* dI2warped,float* du,unsigned int width,
                                      unsigned int height,unsigned int stride,const TooN::Matrix<3,3>& K,
                                      const TooN::Matrix<3,3>& Kinv, const TooN::Matrix<3,3>& R_lr, const TooN::Matrix<3,1>& Kt,
                                             float dmin,
                                             float dmax)
{
    dim3 block(8,8,1);
    dim3 grid(width / block.x, height / block.y, 1);

    cumat3x3   KRKinv = cumat_from<3,3,float>(K*R_lr*Kinv);
    cumat<3,1> cudaKt = cumat_from<3,1,float>(Kt);

    depth_estimation_I2warped<<<grid, block>>>(dI2warped,du,width,
                                               height,stride,KRKinv,cudaKt,
                                               dmin,
                                               dmax);

}



__global__ void depth_estimation_kernel_compute_data_term_and_gradient(const float *dI1, float *data_term,
                                                                   float *grad_wrt_d_at_d0,
                                                                   float *du,float *du0,
                                                                   const unsigned int height,
                                                                   const unsigned int width,
                                                                   const unsigned int stride,
                                                                   const cumat3x3 KRKinv,
                                                                   const cumat<3,1> cudaKt,
                                                                   float dmin,
                                                                   float dmax)
{


    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;


//    float xinterp0 = (float)x+ du0[y*stride+x];
//    float xinterp1 = (float)x+ du0[y*stride+x]+1;


//    float I2_u0      = tex2D(TexImg2,xinterp0+0.5,(float)y+0.5);
//    float I1_val     = dI1[y*stride+x];
//    float grad_I2_u0 = tex2D(TexImg2,xinterp1+0.5,(float)y+0.5) - tex2D(TexImg2,xinterp0+0.5,(float)y+0.5);

//    float u0 = du0[y*stride+x];
//    float u  = du[y*stride+x];

//    grad_wrt_d_at_d0[y*stride+x] = grad_I2_u0;

//    float data_term_value  = (I2_u0 + (u-u0)*grad_I2_u0 - I1_val);

//    data_term[y*stride+x] = data_term_value;


    /// [grad_Ix grad_Iy]*[ 1/w 0 -u/w^2 * KRK^(-1) [x
    ///                    0 1/w -v/w^2]             y
    ///                                              1];

    float uval = (du0[y*stride+x]);//= dmin + du0[y*stride+x]*(dmax-dmin);

//    uval = fminf(1,fmaxf(0.0f,uval));

    const cumat<3,1> p0 = {x*uval,y*uval,uval};
    const cumat<3,1> p2 = KRKinv*p0 + cudaKt;

    const float3 Pc2    = {p2(0,0),p2(1,0),p2(2,0)};
    float2 pIn2   = {p2(0,0)/p2(2,0), p2(1,0)/p2(2,0)};

    pIn2.x = min((float)(width-1), max(0.0f,pIn2.x));
    pIn2.y = min((float)(height-1),max(0.0f,pIn2.y));

    const cumat<3,1> p0withoutdepth = {x,y,1};
    const cumat<3,1> p2withoutdepth = KRKinv*p0withoutdepth;

    /// KRK^(-1)[x y 1]';
    float3 KRKinvpoint = make_float3(p2withoutdepth(0,0), p2withoutdepth(1,0), p2withoutdepth(2,0));

    /// dIdx: Gradient [grad_Ix, grad_Iy]
    float grad_Ix = tex2D(TexImg2,pIn2.x+1+0.5,pIn2.y+0.5) -tex2D(TexImg2,pIn2.x+0.5,pIn2.y+0.5);
    float grad_Iy = tex2D(TexImg2,pIn2.x+0.5,pIn2.y+1+0.5) -tex2D(TexImg2,pIn2.x+0.5,pIn2.y+0.5);

    float2 dIdx = make_float2(grad_Ix,grad_Iy);

    /// grad_pi : Matrix 2x3
    float3 grad_wrt_pi_d_u = make_float3(1/Pc2.z,0,-Pc2.x/(Pc2.z*Pc2.z));
    float3 grad_wrt_pi_d_v = make_float3(0,1/Pc2.z,-Pc2.y/(Pc2.z*Pc2.z));

    /// grad_pi dot KRKinvpoint
    float2 grad_wrt_pi_times_KRKinv = make_float2(dot(grad_wrt_pi_d_u,KRKinvpoint) , dot(grad_wrt_pi_d_v,KRKinvpoint));

    /// dIdd  = dIdx dot (grad_pi dot KRKinv)
    float grad_I2_wrt_d_at_u0 = dot(grad_wrt_pi_times_KRKinv,dIdx);

    grad_I2_wrt_d_at_u0  = grad_I2_wrt_d_at_u0;

    data_term[y*stride+x] = tex2D(TexImg2,pIn2.x+0.5,pIn2.y+0.5)- dI1[y*stride+x] + (du[y*stride+x]-du0[y*stride+x])*grad_I2_wrt_d_at_u0 ;
    grad_wrt_d_at_d0[y*stride+x] = grad_I2_wrt_d_at_u0;

}




void launch_depth_estimation_kernel_compute_data_term_and_gradient(const float *dI1, float *data_term,
                                                                   float *grad_wrt_d_at_d0,
                                                                   float *du,
                                                                   float *du0,
                                                                   const unsigned int height,
                                                                   const unsigned int width,
                                                                   const unsigned int stride,
                                                                   const TooN::Matrix<3,3>& K,
                                                                   const TooN::Matrix<3,3>& Kinv,
                                                                   const TooN::Matrix<3,3>& R_lr,
                                                                   const TooN::Matrix<3,1>& Kt,
                                                                   float dmin,
                                                                   float dmax)
{


    dim3 block(8,8,1);
    dim3 grid(width / block.x, height / block.y, 1);

    cumat3x3   KRKinv = cumat_from<3,3,float>(K*R_lr*Kinv);
    cumat<3,1> cudaKt = cumat_from<3,1,float>(Kt);

//    printf("K(0,0) = %f\n",K(0,0));
//    printf("K(0,1) = %f\n",K(0,1));
//    printf("K(0,2) = %f\n",K(0,2));

//    printf("K(1,0) = %f\n",K(1,0));
//    printf("K(1,1) = %f\n",K(1,1));
//    printf("K(1,2) = %f\n",K(1,2));

//    printf("K(2,0) = %f\n",K(2,0));
//    printf("K(2,1) = %f\n",K(2,1));
//    printf("K(2,2) = %f\n",K(2,2));

//    printf("KRKinv(0,0) = %f\n",KRKinv(0,0));
//    printf("KRKinv(0,1) = %f\n",KRKinv(0,1));
//    printf("KRKinv(0,2) = %f\n",KRKinv(0,2));

//    printf("KRKinv(1,0) = %f\n",KRKinv(1,0));
//    printf("KRKinv(1,1) = %f\n",KRKinv(1,1));
//    printf("KRKinv(1,2) = %f\n",KRKinv(1,2));

//    printf("KRKinv(2,0) = %f\n",KRKinv(2,0));
//    printf("KRKinv(2,1) = %f\n",KRKinv(2,1));
//    printf("KRKinv(2,2) = %f\n",KRKinv(2,2));

//    printf("cudaKt(0,0) = %f\n",cudaKt(0,0));
//    printf("cudaKt(1,0) = %f\n",cudaKt(1,0));
//    printf("cudaKt(2,0) = %f\n",cudaKt(2,0));


    depth_estimation_kernel_compute_data_term_and_gradient<<<grid, block>>>(dI1,data_term,
                                                                            grad_wrt_d_at_d0,
                                                                            du, du0,
                                                                            height,
                                                                            width,
                                                                            stride,
                                                                            KRKinv,
                                                                            cudaKt,
                                                                            dmin,
                                                                            dmax);

}

void launch_depth_kernel(float* dI2, unsigned int width, unsigned int height, unsigned int imgStride)

{
    dim3 block(8,8,1);
    dim3 grid(width / block.x, height / block.y, 1);
    hipBindTexture2D(0,TexImg2,dI2,chandesc_float1,width, height,imgStride*sizeof(float));

    TexImg2.addressMode[0] = hipAddressModeClamp;
    TexImg2.addressMode[1] = hipAddressModeClamp;
    TexImg2.filterMode = hipFilterModeLinear;
    TexImg2.normalized = false;    // access with normalized texture coordinates


}









__global__ void depth_estimation_kernel_check_KRt_is_correct(float *du0,
                                                             unsigned int height,
                                                             unsigned int width,
                                                             unsigned int stride,
                                                             const cumat3x3 KRKinv,
                                                             const cumat<3,1> cudaKt)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float uval = du0[y*stride+x];

    const cumat<3,1> p0 = {x*uval,y*uval,uval};
    const cumat<3,1> p2 = KRKinv*p0 + cudaKt;
    const float2 pIn2   = {p2(0,0)/p2(2,0), p2(1,0)/p2(2,0)};

    if ( x == 112   && y == 112 )
    {
        printf("uval = %f\n",uval);
        printf("%f %f\n",pIn2.x,pIn2.y);
    }

}


void launch_kernel_check_KRt_is_correct(float *du0,
                                        const unsigned int height,
                                        const unsigned int width,
                                        const unsigned int stride,
                                        const TooN::Matrix<3,3>& K,
                                        const TooN::Matrix<3,3>& Kinv,
                                        const TooN::Matrix<3,3>& R_lr,
                                        const TooN::Matrix<3,1>& Kt)

{
    dim3 block(8,8,1);
    dim3 grid(width / block.x, height / block.y, 1);

    cumat3x3   KRKinv = cumat_from<3,3,float>(K*R_lr*Kinv);
    cumat<3,1> cudaKt = cumat_from<3,1,float>(Kt);

    depth_estimation_kernel_check_KRt_is_correct<<<grid,block>>>(du0, height,
                                                                 width,stride,
                                                                 KRKinv,
                                                                 cudaKt);
}


__global__ void depth_estimation_kernel_check_grad_wrt_d0_is_correct(float *du0,
                                                                     const unsigned int height,
                                                                     const unsigned int width,
                                                                     const unsigned int stride,
                                                                     const cumat3x3 KRKinv,
                                                                     const cumat<3,1> cudaKt)
{


    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    const cumat<3,1> p0 = {x*du0[y*stride+x],y*du0[y*stride+x],du0[y*stride+x]};
    const cumat<3,1> p2 = KRKinv*p0 + cudaKt;
    const float2 pIn2   = {p2(0,0)/p2(2,0), p2(1,0)/p2(2,0)};

    const cumat<3,1> p0withoutdepth = {x,y,1};
    const cumat<3,1> p2withoutdepth = KRKinv*p0withoutdepth;

    float grad_Ix = tex2D(TexImg2,pIn2.x+1+0.5,pIn2.y+0.5) -tex2D(TexImg2,pIn2.x+0.5,pIn2.y+0.5);
    float grad_Iy = tex2D(TexImg2,pIn2.x+0.5,pIn2.y+1+0.5) -tex2D(TexImg2,pIn2.x+0.5,pIn2.y+0.5);

    float p2_wsqr     = p2(2,0)*p2(2,0);
    float third_val = -(1/p2_wsqr)*(p2(0,0)*grad_Ix + p2(1,0)*grad_Iy);

    float3 grad_wrt_pi   = {grad_Ix*(1/p2(2,0)), grad_Iy*(1/p2(2,0)), third_val};
    float3 pxyz          = {p2withoutdepth(0,0), p2withoutdepth(1,0), p2withoutdepth(2,0)};

    float grad_I2_wrt_d_at_u0 = dot(grad_wrt_pi,pxyz);

    if ( x == 123 && y == 165 )
    {
        printf("d0 = %f\n",du0[y*stride+x]);

        printf("pIn2 = %f %f\n",pIn2.x,pIn2.y);

        printf("p2withoutdepth = %f %f %f\n",p2withoutdepth(0,0),p2withoutdepth(1,0),p2withoutdepth(2,0));

        printf("grad_val = %f\n",grad_I2_wrt_d_at_u0);
    }

}


void launch_kernel_check_grad_wrt_d0_is_correct(float *du0,
                                        const unsigned int height,
                                        const unsigned int width,
                                        const unsigned int stride,
                                        const TooN::Matrix<3,3>& K,
                                        const TooN::Matrix<3,3>& Kinv,
                                        const TooN::Matrix<3,3>& R_lr,
                                        const TooN::Matrix<3,1>& Kt)

{
    dim3 block(8,8,1);
    dim3 grid(width / block.x, height / block.y, 1);

    cumat3x3   KRKinv = cumat_from<3,3,float>(K*R_lr*Kinv);
    cumat<3,1> cudaKt = cumat_from<3,1,float>(Kt);

    depth_estimation_kernel_check_grad_wrt_d0_is_correct<<<grid,block>>>(du0, height,
                                                                 width,stride,
                                                                 KRKinv,
                                                                 cudaKt);
}



