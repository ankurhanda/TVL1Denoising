#include "hip/hip_runtime.h"
/**
 * @author  Ankur Handa
 * Copyright (C) 2011  Ankur Handa
 *                     Imperial College London
 **/

#ifndef _SIMPLEGL_KERNEL_H_
#define _SIMPLEGL_KERNEL_H_

#include <stdio.h>
#include <cutil_inline.h>

texture<float, 2, hipReadModeElementType> my_tex;


const static hipChannelFormatDesc chandesc_float1 =
hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif





__global__ void kernel_dualp(float *px, float *py, float *ux, float *uy, float sigma, unsigned int stride, unsigned int width, unsigned int height)
{


    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // write output vertex
    px[y*stride+x] = px[y*stride+x] + sigma* ux[y*stride+x];
    py[y*stride+x] = py[y*stride+x] + sigma* uy[y*stride+x];

    float pxval = px[y*stride+x];
    float pyval = py[y*stride+x];

    float reprojection = 0;
    reprojection   = sqrt(pxval*pxval + pyval*pyval);
    reprojection   = max(1,reprojection);

    px[y*stride+x] = px[y*stride+x]/reprojection;
    py[y*stride+x] = py[y*stride+x]/reprojection;


}


__global__ void kernel_dualq(float *dq, float *u, float* g, float sigma, float lambda, unsigned int stride, unsigned int width, unsigned int height)
{


    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // write output vertex
    dq[y*stride+x] = dq[y*stride+x] + sigma*lambda*(u[y*stride+x] - g[y*stride+x]);

    float reprojection = 0;
    reprojection   = fabs(dq[y*stride+x]);
    reprojection   = max(1,reprojection);

    dq[y*stride+x] = dq[y*stride+x]/reprojection;


}


__global__ void kernel_update_u(float *px, float *py, float *u, float* dq, unsigned int stride, unsigned int width, unsigned int height, float tau, float lambda)
{

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float dxp = 0 , dyp = 0;

    if ( x >= 1 && x < width )  dxp = px[y*stride+x] - px[y*stride+(x-1)];

    if ( y >= 1 && y < height ) dyp = py[y*stride+x] - py[(y-1)*stride+x];

    float divp = dxp + dyp;

    u[y*stride+x] = (u[y*stride+x] + tau*(divp - lambda*dq[y*stride+x]));


}


__global__ void kernel_derivative_u(float *ux, float *uy, float *u, unsigned int stride, unsigned int width, unsigned int height )
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if ( x + 1 < width ) //&& y < height )
    {
        ux[y*stride+x] = u[y*stride+(x+1)] - u[y*stride+x];
    }

    if ( y + 1 < height )
    {
        uy[y*stride+x] = u[(y+1)*stride+x] - u[y*stride+x];
    }

}

//__global__ void kernel_disparity_estimation(float *du, float *du0, float *dp,
//                                            float *dq, float *dI1, float sigma_q,
//                                            float sigma_p, float sigma_u, float lambda)
//{

//    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
//    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

//    float I2_u0 = tex2D(my_tex,);

//    float data_term = lambda*(I2_u0 + (u-u0)*grad_I2_u0 - I1);

//    dq[y*stride+u] = dq[y*stride+x] + sigma_q*(data_term);



//}


extern "C" void launch_kernel_derivative_u(float* ux, float *uy, float* u, unsigned int stride, unsigned int mesh_width, unsigned int mesh_height)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);


    kernel_derivative_u<<< grid, block>>>(ux, uy, u, stride, mesh_width, mesh_height);


    cutilCheckMsg("execution failed\n");


}


extern "C" void launch_kernel_update_u(float *px, float *py, float *u, float* dq, unsigned int stride, unsigned int mesh_width, unsigned int mesh_height, float tau, float lambda)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    kernel_update_u<<< grid, block>>>(px,py,u,dq, stride, mesh_width, mesh_height, tau, lambda);
    cutilCheckMsg("execution failed\n");
}


// Wrapper for the __global__ call that sets up the kernel call
extern "C" void launch_kernel_dual_variable_p(float *px, float *py, float* ux, float *uy, float sigma, unsigned int stride, unsigned int mesh_width, unsigned int mesh_height)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    kernel_dualp<<< grid, block>>>(px,py,ux,uy,sigma, stride, mesh_width, mesh_height);
    cutilCheckMsg("execution failed\n");
}


extern "C" void launch_kernel_dual_variable_q(float *dq, float *u, float *g, float sigma, float lambda, unsigned int stride, unsigned int mesh_width, unsigned int mesh_height)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    kernel_dualq<<< grid, block>>>(dq,u,g,sigma, lambda, stride, mesh_width, mesh_height);
    cutilCheckMsg("execution failed\n");
}




#endif // #ifndef _SIMPLEGL_KERNEL_H_
